#include "hip/hip_runtime.h"
// =================================================================
//
// File: example01b.cpp
// Author: Pedro Perez
// Description: This file implements the addition of two vectors. 
//				The time this implementation takes will be used as 
//				the basis to calculate the improvement obtained with 
//				parallel technologies.
//
// Copyright (c) 2024 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <iostream>
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>
#include "utils.h"

using namespace std;
using namespace std::chrono;

#define SIZE 1000 // 1e9

__global__ void add_vector(int *result, int *a, int *b) {
    result[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(int argc, char* argv[]) {
    int *a, *b, *c;
    int *deviceA, *deviceB, *deviceC;

    // These variables are used to keep track of the execution time.
    high_resolution_clock::time_point start, end;
    double timeElapsed;

    a = new int [SIZE];
    b = new int [SIZE];
    c = new int [SIZE];

    hipMalloc((void**) &deviceA, SIZE * sizeof(int));
    hipMalloc((void**) &deviceB, SIZE * sizeof(int));
    hipMalloc((void**) &deviceC, SIZE * sizeof(int));

    fill_array(a, SIZE);
    display_array("a:", a);
    fill_array(b, SIZE);
    display_array("b:", b);

    hipMemcpy(deviceA, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

    cout << "Starting...\n";
    timeElapsed = 0;
    for (int j = 0; j < N; j++) {
        start = high_resolution_clock::now();

        add_vector<<<1, SIZE>>>(deviceC, deviceA, deviceB);

        end = high_resolution_clock::now();
        timeElapsed += 
            duration<double, std::milli>(end - start).count();
    }
    hipMemcpy(c, deviceC, SIZE * sizeof(int), hipMemcpyDeviceToHost);
    display_array("c:", c);
    cout << "avg time = " << fixed << setprecision(3) 
         << (timeElapsed / N) <<  " ms\n";

    delete [] a;
    delete [] b;
    delete [] c;

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}
