// =================================================================
//
// File: intro01.cu
// Author: Pedro Perez
// Description: This file shows some of the basic CUDA directives.
//		        To compile: nvcc -o app intro02.cu
//
// Copyright (c) 2023 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main(int argc, char* argv[]) {
    int a, b, c;
    int *d_a, *d_b, *d_c;
    
    a = 12;
    b = 13;

    hipMalloc((void**) &d_a, sizeof(int));
    hipMalloc((void**) &d_b, sizeof(int));
    hipMalloc((void**) &d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    add<<<1, 1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    cout << "c = " << c << "\n";

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
